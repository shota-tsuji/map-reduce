#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define CHECK(call)																									\
{																																		\
	const hipError_t error = call;																		\
	if (error != hipSuccess)																					\
	{																																	\
		printf("Error: %s:%d, ", __FILE__, __LINE__);										\
		printf("code:%d, reason: %s\n", error,													\
						hipGetErrorString(error));															\
		exit(1);																												\
	}																																	\
}

double cpuSecond() {
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
	double epsilon = 1.0E-8;
	bool match = 1;

	for (int i=0; i<N; i++) {
		if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
			match = 0;
			printf("Arrays do not match!\n");
			printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
																											gpuRef[i], i);
			break;
		}
	}

	if (match) printf("Arrays match.\n\n");

	return;
}

void initialData(float *ip, int size){
	time_t t; // random value
	srand((unsigned int ) time(&t));

	for (int i=0; i<size; i++){
		ip[i] = (float)( rand() & 0xFF  ) / 10.0f;
		return;
	}
}

void sumArrayOnHost(float *A, float *B, float *C, const int N){
	for (int idx=0; idx<N; idx++) {
		C[idx] = A[idx] + B[idx];
	}
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
	//int i = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//int i = blockIdx.x;
	C[i] = A[i] + B[i];
}

int main(int argc, char **argv) {
	printf("%s Starting...\n", argv[0]);

	// デバイスのセットアップ
	int dev = 0;
	//hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// ベクトルのデータサイズを設定
	int nElem = 1 << 24;
	printf("Vector size %d\n", nElem);

	// ホストメモリを確保
	size_t nBytes = nElem * sizeof(float);

	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	hostRef = (float *)malloc(nBytes);
	gpuRef = (float *)malloc(nBytes);

	double iStart, iElaps;

	// ホスト側でデータを初期化
	iStart = cpuSecond();
	initialData(h_A, nElem);
	initialData(h_B, nElem);
	iElaps = cpuSecond() - iStart;
	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);

	// 結果をチェックするためにホスト側でベクトルを加算
	iStart = cpuSecond();
	sumArrayOnHost(h_A, h_B, hostRef, nElem);
	iElaps = cpuSecond() - iStart;

	// デバイスのグローバルメモリを確保
	float *d_A, *d_B, *d_C;
	CHECK(hipMalloc((float**)&d_A, nBytes));
	CHECK(hipMalloc((float**)&d_B, nBytes));
	CHECK(hipMalloc((float**)&d_C, nBytes));
	

	// ホストからデバイスへデータを転送
	CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));	
	CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));
	

	// ホスト側でカーネルを呼び出す
	int iLen = 1024;
	dim3 block(iLen);
	dim3 grid((nElem + block.x -1) / block.x);

	iStart = cpuSecond();
	sumArraysOnGPU<<< grid, block >>>(d_A, d_B, d_C, nElem);
	CHECK(hipDeviceSynchronize());
	iElaps = cpuSecond() - iStart;
	printf("sumArraysOnGPU <<<%d, %d>>> Time elapsed %f" \
					"sec\n", grid.x, block.x, iElaps);

	// カーネルのエラーをチェック
	CHECK(hipGetLastError());

	// カーネルの結果をホスト側にコピー
	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

	// デバイスの結果をチェック
	checkResult(hostRef, gpuRef, nElem);

	// デバイスのグローバルメモリを解放
	CHECK(hipFree(d_A));
	CHECK(hipFree(d_B));
	CHECK(hipFree(d_C));

	// ホストのメモリを解放
	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);

	hipDeviceReset();
	return(0);
}
