#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define CHECK(call)																									\
{																																		\
	const hipError_t error = call;																		\
	if (error != hipSuccess)																					\
	{																																	\
		printf("Error: %s:%d, ", __FILE__, __LINE__);										\
		printf("code:%d, reason: %s\n", error,													\
						hipGetErrorString(error));															\
		exit(1);																												\
	}																																	\
}

double cpuSecond() {
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
	double epsilon = 1.0E-8;
	bool match = 1;

	for (int i=0; i<N; i++) {
		if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
			match = 0;
			printf("Arrays do not match!\n");
			printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
																											gpuRef[i], i);
			break;
		}
	}

	if (match) printf("Arrays match.\n\n");

	return;
}

void initialData(float *ip, int size){
	time_t t; // random value
	srand((unsigned int ) time(&t));

	for (int i=0; i<size; i++){
		ip[i] = (float)( rand() & 0xFF  ) / 10.0f;
		return;
	}
}

void sumArrayOnHost(float *A, float *B, float *C, const int N){
	for (int idx=0; idx<N; idx++) {
		C[idx] = A[idx] + B[idx];
	}
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
	//int i = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//int i = blockIdx.x;
	C[i] = A[i] + B[i];
}

int main(int argc, char **argv) {
	printf("%s Starting...\n", argv[0]);

	// デバイスのセットアップ
	int dev = 0;
	//hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceP.name);
	CHECK(hipSetDevice(dev));

	// ベクトルのデータサイズを設定
	int nElem = 1 << 24;
	printf("Vector size %d\n", nElem);

	// ホストメモリを確保
	size_t nBytes = nElem * sizeof(float);

	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	hostRef = (float *)malloc(nBytes);
	gpuRef = (float *)malloc(nBytes);

	double iStart, iElaps;

	// ホスト側でデータを初期化
	initialData(h_A, nElem);
	initialData(h_B, nElem);

	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);

	// デバイスのグローバルメモリを確保
	float *d_A, *d_B, *d_C;
	hipMalloc((float**)&d_A, nBytes);
	hipMalloc((float**)&d_B, nBytes);
	hipMalloc((float**)&d_C, nBytes);

	// ホストからデバイスへデータを転送
	hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice);

	// ホスト側でカーネルを呼び出す
	dim3 block(nElem);
	dim3 grid(1);
	//dim3 block(1);
	//dim3 grid(32);

	sumArraysOnGPU<<< grid, block >>>(d_A, d_B, d_C, nElem);
	printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);

	// カーネルの結果をホスト側にコピー
	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

	// 結果をチェックするためにホスト側でベクトルを加算
	sumArrayOnHost(h_A, h_B, hostRef, nElem);

	// デバイスの結果をチェック
	checkResult(hostRef, gpuRef, nElem);

	// デバイスのグローバルメモリを解放
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// ホストのメモリを解放
	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);

	hipDeviceReset();
	return(0);
}
