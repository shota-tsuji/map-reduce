#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv) {
	//データ要素の合計数を定義
	int nElem = 1024;

	// グリッドとブロックの構造を定義
	dim3 block(1024);
	dim3 grid((nElem + block.x -1) / block.x);
	printf("grid.x %d block.x %d \n", grid.x, block.x);

	// ブロックをリセット
	block.x = 512;
	grid.x = (nElem + block.x -1) / block.x;
	printf("grid.x %d block.x %d \n", grid.x, block.x);

	// ブロックをリセット
	block.x = 256;
	grid.x = (nElem + block.x -1) / block.x;
	printf("grid.x %d block.x %d \n", grid.x, block.x);

	// ブロックをリセット
	block.x = 128;
	grid.x = (nElem + block.x -1) / block.x;
	printf("grid.x %d block.x %d \n", grid.x, block.x);

	// デバイスをリセット
	hipDeviceReset();

	return (0);
}
